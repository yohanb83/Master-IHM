


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BSIZE 1024

__global__ void kvecsum1(unsigned* vec, int n){
    int idX = blockIdx.x*blockDim.x + threadIdx.x;
    int offset;
    for(offset=n/2; offset>0; offset/=2){
        if(idX<offset){
            vec[idX] += vec[idX+offset];
        }
        __syncthreads();
    }
}

void vecsum1(unsigned* vec, unsigned* sum, int size){
    unsigned* d_vec;
    int bytes = size*sizeof(unsigned);

    hipMalloc((void**) &d_vec, bytes);
    hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
    kvecsum1<<<1, size>>>(d_vec, size);
    hipMemcpy(sum, d_vec, sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_vec);
}

void vecsum2(unsigned* vec, unsigned* sum, int size){
    unsigned *d_vec, *d_sum;
    int vec_bytes = size*sizeof(unsigned);
    int sum_bytes = (size/BSIZE)*sizeof(unsigned);

    hipMalloc((void**) &d_vec, vec_bytes);
    hipMalloc((void**) &d_sum, sum_bytes);

    hipMemcpy(d_vec, vec, vec_bytes, hipMemcpyHostToDevice);
    kvecsum1<<<1, size>>>(d_vec, size);
    hipMemcpy(sum, d_vec, sizeof(unsigned), hipMemcpyDeviceToHost);

    hipMemcpy(d_sum, sum, sum_bytes, hipMemcpyHostToDevice);
    kvecsum1<<<1, size/BSIZE>>>(d_sum, size/BSIZE);
    hipMemcpy(sum, d_sum, sizeof(unsigned), hipMemcpyDeviceToHost);

    hipFree(d_vec); hipFree(d_sum);
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: <filename>\n");
        exit(-1);
    }
    unsigned int log2size, size;
    unsigned int *vec;
    FILE *f = fopen(argv[1], "r");
    fscanf(f, "%d\n", &log2size);
    if (log2size > 10) {
        printf("Size (%u) is too large: size is limited to 2^10\n", log2size);
        exit(-1);
    }
    size = 1 << log2size;
    unsigned int bytes = size * sizeof(unsigned int);
    vec = (unsigned int *) malloc(bytes);
    assert(vec);
    for (unsigned int i = 0; i < size; i++) {
        fscanf(f, "%u\n", &(vec[i]));
    }
    fclose(f);

    unsigned sum[1] = {0};
    vecsum1(vec, sum, size);
    printf("%u\n", sum[0]);

}
