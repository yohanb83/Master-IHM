

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kvecsum1(unsigned* vec, int n){
    int idX = threadIdx.x;
    int offset;
    for(offset=n/2; offset>0; offset/=2){
        if(idX<offset){
            vec[idX] += vec[idX+offset];
        }
        __syncthreads();
    }

}

void vecsum1(unsigned* vec, unsigned* sum, int size){
    unsigned* d_vec;
    int bytes = size*sizeof(unsigned);

    hipMalloc((void**) &d_vec, bytes);
    hipMemcpy(d_vec, vec, bytes, hipMemcpyHostToDevice);
    kvecsum1<<<1, size>>>(d_vec, size);
    hipMemcpy(sum, d_vec, sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_vec);
}


int main(void){
    exit(0);
}
